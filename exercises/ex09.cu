#include "hip/hip_runtime.h"
// CUDA programming
// Exercise n. 09

#include <errno.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKS  2
#define THREADS 2

// Prototypes
__global__ void square_matrix_transpose(int *d_X, int *d_Y, int N);
__host__ void initialize_array(int *array, int N)
__host__ void print_matrix(int *A, int N);

int main(void)
{
    int *A, *B;        // host copies of A, B
    int *d_A, *d_B;    // device copies of A, B

    int    N = BLOCKS * THREADS;
    int size = N * N * sizeof(int);

    // Allocate space for host copies of A, B
    A = (int *)malloc(size);
    B = (int *)malloc(size);

    // Setup input values
    initialize_array(A, N * N);

    // Allocate space for device copies of A, B
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);

    // Copy inputs to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Setup the execution configuration
    dim3 dim_grid(BLOCKS, BLOCKS, 1);     // size: BLOCKS x BLOCKS x 1
    dim3 dim_block(THREADS, THREADS, 1);  // size: THREADS x THREADS x 1

    // Call the kernel on GPU
    square_matrix_transpose<<< dim_grid, dim_block >>>(d_A, d_B, N);

    // Copy result back to host
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

    // Check the result
    print_matrix(A, N);
    print_matrix(B, N);

    // Cleanup
    free(A);
    free(B);
    hipFree(d_A);
    hipFree(d_B);

    return(EXIT_SUCCESS);
}

// Transpose of a square matrix (on device)
__global__ void square_matrix_transpose(int *d_X, int *d_Y, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid accessing beyond the end of the matrices
    if(row < N && col < N)
    {
        for(int k = 0; k < N; k++)
        {
            unsigned int    pos = row * N + col;
            unsigned int tr_pos = col * N + row;
            d_Y[tr_pos] = d_X[pos];
        }
    }
}

// Host function to initialize an array
__host__ void initialize_array(int *array, int N)
{
    for (int i = 0; i < N; i++)
    {
        array[i] = i + 1;  // Sequential integers
    }
}

// Host function to print a matrix
__host__ void print_matrix(int *A, int N)
{
    for(int i = 0; i < N; i++)
    {
        for(int j = 0; j < N; j++)
        {
            printf("%d\t", A[i * N + j]);
        }
        printf("\n");
    }
    printf("\n");
}
