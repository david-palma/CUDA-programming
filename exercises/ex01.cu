// CUDA programming
// Exercise n. 0

#include <errno.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

// Prototype
__host__ void print_dev_prop(hipDeviceProp_t dev_prop);

int main(void)
{
    // Number of CUDA-capable devices attached to this system
    int dev_cnt;
    hipGetDeviceCount(&dev_cnt);

    // Calculate the theoretical peak bandwidth for each device
    for(int i = 0; i < dev_cnt; i++)
    {
        hipDeviceProp_t dev_prop;
        hipGetDeviceProperties(&dev_prop, i);
        printf("Device Number: %d\n", i);
        print_dev_prop(dev_prop);
    }
  }

// Print device properties
__host__ void print_dev_prop(hipDeviceProp_t dev_prop)
{
    printf("  Major revision number:         %d\n",  dev_prop.major);
    printf("  Minor revision number:         %d\n",  dev_prop.minor);
    printf("  Name:                          %s\n",  dev_prop.name);
    printf("  Total global memory:           %zu\n", dev_prop.totalGlobalMem);
    printf("  Total shared memory per block: %zu\n", dev_prop.sharedMemPerBlock);
    printf("  Total registers per block:     %d\n",  dev_prop.regsPerBlock);
    printf("  Warp size:                     %d\n",  dev_prop.warpSize);
    printf("  Maximum memory pitch:          %zu\n", dev_prop.memPitch);
    printf("  Maximum threads per block:     %d\n",  dev_prop.maxThreadsPerBlock);

    for(int i = 0; i < 3; ++i)
        printf("  Maximum block dimension #%02d:   %d\n", i, dev_prop.maxThreadsDim[i]);

    for(int i = 0; i < 3; ++i)
        printf("  Maximum grid dimension #%02d:    %d\n", i, dev_prop.maxGridSize[i]);

    printf("  Clock rate:                    %d\n",  dev_prop.clockRate);
    printf("  Memory Bus Width (bits):       %d\n",  dev_prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s):  %f\n\n", 2.0 * dev_prop.memoryClockRate * (dev_prop.memoryBusWidth / 8) / 1.0e6);
    printf("  Total constant memory:         %zu\n", dev_prop.totalConstMem);
    printf("  Texture alignment:             %zu\n", dev_prop.textureAlignment);
    printf("  Concurrent copy and execution: %s\n", (dev_prop.deviceOverlap ? "Yes" : "No"));
    printf("  Number of multiprocessors:     %d\n",  dev_prop.multiProcessorCount);
    printf("  Kernel execution timeout:      %s\n", (dev_prop.kernelExecTimeoutEnabled ? "Yes" : "No"));

   return;
}
