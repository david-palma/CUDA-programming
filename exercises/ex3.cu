// CUDA programming
// Exercise n. 3

#include <errno.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKS  8
#define THREADS 1

// Prototype
__global__ void add(int *a, int *b, int *c);
__host__ void ints(int *m, int N);
__host__ void print_array(int *a, int N);

int main(void)
{
    int *a, *b, *c;         // host copies of a, b, c
    int *d_a, *d_b, *d_c;   // device copies of a, b, c

    int    N = BLOCKS * THREADS;
    int size = N * sizeof(int);

    // Allocate space for host copies of a, b, c
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Setup input values
    ints(a, N);
    ints(b, N);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Call the kernel on GPU
    add<<< BLOCKS, THREADS >>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Check the result
    print_array(a, N);
    print_array(b, N);
    print_array(c, N);

    // Cleanup
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return(EXIT_SUCCESS);
}

// Vector addition
__global__ void add(int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

// Initialisation
__host__ void ints(int *m, int N)
{
    int i;
    for(i = 0; i < N; i++)
        m[i] = i;
}

// Print the elements of the array
__host__ void print_array(int *a, int N)
{
    for(int i = 0; i < N; i++)
    {
        printf("%d\t", a[i]);
    }
    printf("\n");
}
