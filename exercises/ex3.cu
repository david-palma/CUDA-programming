#include "hip/hip_runtime.h"
// CUDA programming
// Exercise n. 3

#include <errno.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKS  512
#define THREADS 1

// Prototype
__global__ add(int *a, int *b, int *c);

int main(void)
{
    int *a, *b, *c;         // host copies of a, b, c
    int *d_a, *d_b, *d_c;   // device copies of a, b, c

    int    N = BLOCKS * THREADS;
    int size = N * sizeof(int);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Allocate space for device copies of a, b, c
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Setup input values
    random_ints(a, N);
    random_ints(b, N);

    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Call the add() kernel on GPU
    add<<< BLOCKS, THREADS >>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return(EXIT_SUCCESS);
}

// Vector addition
__global__ add(int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}
