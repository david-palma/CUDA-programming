#include "hip/hip_runtime.h"
// CUDA programming
// Exercise n. 3

#include <errno.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKS  512
#define THREADS   1

// Prototype
__global__ add(int *a, int *b, int *c);
__host__ void ints(int* m, int N);

int main(void)
{
    int *a, *b, *c;         // host copies of a, b, c
    int *d_a, *d_b, *d_c;   // device copies of a, b, c

    int    N = BLOCKS * THREADS;
    int size = N * sizeof(int);

    // Allocate space for host copies of a, b, c
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Setup input values
    ints(a, N);
    ints(b, N);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Call the add() kernel on GPU
    add<<< BLOCKS, THREADS >>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return(EXIT_SUCCESS);
}

// Vector addition
__global__ add(int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

// Initialisation
__host__ void ints(int* m, int N)
{
    int i;
    for (i = 0; i < N; i++)
        m[i] = i;
}
