// CUDA programming
// Exercise n. 10

#include <errno.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N_ELEMS 16
#define THREADS  4

// Prototype
__global__ void dot_prod(int *a, int *b, int *c);
__host__ void ints(int *m, int N);
__host__ void print_array(int *a, int N);

int main(void)
{
    int *a, *b, *c;         // host copies of a, b, c
    int *d_a, *d_b, *d_c;   // device copies of a, b, c
    int   size = N_ELEMS * sizeof(int);

    // Allocate space for host copies of a, b, c
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(sizeof(int));

    // Setup input values
    ints(a, N_ELEMS);
    ints(b, N_ELEMS);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Call the kernel on GPU
    dot_prod<<< N_ELEMS/THREADS, THREADS >>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    // Check the result
    print_array(a, N_ELEMS);
    print_array(b, N_ELEMS);
    printf("%d\n", *c);

    // Cleanup
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return(EXIT_SUCCESS);
}

// Vector addition (on device)
__global__ void dot_prod(int *a, int *b, int *c)
{
    __shared__ int tmp[THREADS];
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    tmp[threadIdx.x] = a[index] * b[index];

    __syncthreads();

    if(0 == threadIdx.x)
    {
        int sum = 0;
        for(int i = 0; i < THREADS; i++)
        {
            sum += tmp[i];
        }
        atomicAdd(c, sum);  // atomic operation to avoid race condition
    }
}

// Initialisation
__host__ void ints(int *m, int N)
{
    int i;
    for(i = 0; i < N; i++)
        m[i] = 1;
}

// Print the elements of the array
__host__ void print_array(int *a, int N)
{
    for(int i = 0; i < N; i++)
    {
        printf("%d\t", a[i]);
    }
    printf("\n");
}
