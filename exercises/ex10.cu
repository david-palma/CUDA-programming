#include "hip/hip_runtime.h"
// CUDA programming
// Exercise n. 10

#include <errno.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N_ELEMS 16
#define THREADS  4

// Prototype
__global__ void dot_prod(int *a, int *b, int *c);
__host__ void initialize_array(int *array, int N)
__host__ void print_array(int *array, int N)

int main(void)
{
    int *a, *b, *c;         // host copies of a, b, c
    int *d_a, *d_b, *d_c;   // device copies of a, b, c
    int   size = N_ELEMS * sizeof(int);

    // Allocate space for host copies of a, b, c
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(sizeof(int));

    // Setup input values
    initialize_array(a, N_ELEMS);
    initialize_array(b, N_ELEMS);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Call the kernel on GPU
    dot_prod<<< N_ELEMS/THREADS, THREADS >>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    // Check the result
    print_array(a, N_ELEMS);
    print_array(b, N_ELEMS);
    printf("%d\n", *c);

    // Cleanup
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return(EXIT_SUCCESS);
}

// Vector addition (on device)
__global__ void dot_prod(int *a, int *b, int *c)
{
    __shared__ int tmp[THREADS];
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    tmp[threadIdx.x] = a[index] * b[index];

    __syncthreads();

    if(0 == threadIdx.x)
    {
        int sum = 0;
        for(int i = 0; i < THREADS; i++)
        {
            sum += tmp[i];
        }
        atomicAdd(c, sum);  // atomic operation to avoid race condition
    }
}

// Host function to initialize an array
__host__ void initialize_array(int *array, int N)
{
    for (int i = 0; i < N; i++)
    {
        array[i] = 1;
    }
}

// Host function to print an array
__host__ void print_array(int *array, int N)
{
    for (int i = 0; i < N; i++)
    {
        printf("%d\t", array[i]);
    }
    printf("\n");
}
