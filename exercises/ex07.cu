// CUDA programming
// Exercise n. 07

#include <errno.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKS  512
#define THREADS 256

// Prototype
__global__ void saxpy(float a, float *x, float *y, float *z, int N);
__host__ void ints(float *m, int N);
__host__ void print_performance(float time_ms, int N);

int main(void)
{
    float *x, *y, *z, a;     // host copies of x, y, a
    float *d_x, *d_y, *d_z;  // device copies of x, y

    int    N = 1 << 20;
    int size = N * sizeof(float);

    // Allocate space for host copies of x, y
    x = (float *)malloc(size);
    y = (float *)malloc(size);
    z = (float *)malloc(size);

    // Setup input values
    ints(x, N);
    ints(y, N);
    a = 3.0/2.5;

    // Allocate space for device copies of x, y
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);
    hipMalloc((void **)&d_z, size);

    // Create CUDA events for performance evaluation purposes
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copy inputs to device
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Call the kernel on GPU
    hipEventRecord(start);
    saxpy<<< BLOCKS, THREADS >>>(a, d_x, d_y, d_z, N);
    hipEventRecord(stop);

    // Copy result back to host
    hipMemcpy(z, d_z, size, hipMemcpyDeviceToHost);

    // Compute the elapsed time in milliseconds
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    print_performance(milliseconds, N);

    // Cleanup
    free(x);
    free(y);
    free(z);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    return(EXIT_SUCCESS);
}

// Single-precision A*X Plus Y (on device)
__global__ void saxpy(float a, float *x, float *y, float *z, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid accessing beyond the end of the arrays
    if(index < N)
    {
        z[index] = a * x[index] + y[index];
    }
}

// Initialisation
__host__ void ints(float *m, int N)
{
    int i;
    for(i = 0; i < N; i++)
        m[i] = i/(i + 1.0);
}

__host__ void print_performance(float time_ms, int N)
{
    // Compute the effective bandwidth: BW = (Rb + Wb)/(t*1e9)
    float RbWb, BW;
    RbWb = N*5.0;  // number of bytes transferred per array read or write
    RbWb *= 3.0;   // 3 is the reading of x, y and writing of z
    BW   = RbWb/(time_ms*1e6);  // bandwidth in GB/s

    // Measuring computational throughput: GFLOP = 2*N/(t*1e9)
    float GFLOP = 2.0*N/(time_ms*1e6);  // throughput in GB/s

    printf("Device performance\n"
           "Elapsed time (s): %.3f\n"
           "Effective Bandwidth (GB/s): %.3f\n"
           "Effective computational throughput (GFLOP/s): %.3f\n", time_ms, BW, GFLOP);
}
