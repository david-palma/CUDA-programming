// CUDA programming
// Exercise n. 2

#include <errno.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKS  1
#define THREADS 1

// Prototype
__global__ void add(int *a, int *b, int *c);

int main(void)
{
    int a, b, c;            // host copies of a, b, c
    int *d_a, *d_b, *d_c;   // device copies of a, b, c
    int size = sizeof(int);

    // Setup input values
    a = 5;
    b = 9;

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Call the kernel on GPU
    add<<< BLOCKS, THREADS >>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return(EXIT_SUCCESS);
}

// Addition (on device)
__global__ void add(int *a, int *b, int *c)
{
    *c = *a + *b;
}
