#include "hip/hip_runtime.h"
// CUDA programming
// Exercise n. 4

#include <errno.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKS  1
#define THREADS 512

// Prototype
__global__ add(int *a, int *b, int *c);

int main(void)
{
    int *a, *b, *c;         // host copies of a, b, c
    int *d_a, *d_b, *d_c;   // device copies of a, b, c

    int    N = BLOCKS * THREADS;
    int size = N * sizeof(int);

    // Allocate space for host copies of a, b, c
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Setup input values
    random_ints(a, N);
    random_ints(b, N);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Call the add() kernel on GPU
    add<<< BLOCKS, THREADS >>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return(EXIT_SUCCESS);
}

// Vector addition
__global__ add(int *a, int *b, int *c)
{
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}
