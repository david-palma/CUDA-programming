#include "hip/hip_runtime.h"
// CUDA programming
// Exercise n. 4

#include <errno.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKS    1
#define THREADS 512

// Prototype
__global__ add(int *a, int *b, int *c);
__host__ void ints(int* m, int N);

int main(void)
{
    int *a, *b, *c;         // host copies of a, b, c
    int *d_a, *d_b, *d_c;   // device copies of a, b, c

    int    N = BLOCKS * THREADS;
    int size = N * sizeof(int);

    // Allocate space for host copies of a, b, c
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Setup input values
    ints(a, N);
    ints(b, N);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Call the add() kernel on GPU
    add<<< BLOCKS, THREADS >>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return(EXIT_SUCCESS);
}

// Vector addition
__global__ add(int *a, int *b, int *c)
{
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

// Initialisation
__host__ void ints(int* m, int N)
{
    int i;
    for (i = 0; i < N; ++i)
        m[i] = i;
}
