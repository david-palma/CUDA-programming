// CUDA programming
// Exercise n. 6

#include <errno.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKS  32
#define THREADS 32

// Prototype
__global__ void saxpy(float a, float *x, float *y, int N);
__host__ void ints(float *m, int N);

int main(void)
{
    float *x, *y, a;    // host copies of x, y, a
    float *d_x, *d_y;   // device copies of x, y

    int    N = 2 * BLOCKS * THREADS;
    int size = N * sizeof(float);

    // Allocate space for host copies of x, y
    x = (float *)malloc(size);
    y = (float *)malloc(size);

    // Setup input values
    ints(x, N);
    ints(y, N);
    a = 3.0;

    // Allocate space for device copies of x, y
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);

    // Copy inputs to device
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Call the kernel on GPU
    saxpy<<< BLOCKS, THREADS >>>(a, d_x, d_y, N);

    // Copy result back to host
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Cleanup
    free(x);
    free(y);
    hipFree(d_x);
    hipFree(d_y);

    return(EXIT_SUCCESS);
}

// Single-precision A*X Plus Y
__global__ void saxpy(float a, float *x, float *y, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid accessing beyond the end of the arrays
    if(index < N)
    {
        y[index] = a * x[index] + y[index];
    }
}

// Initialisation
__host__ void ints(float *m, int N)
{
    int i;
    for(i = 0; i < N; i++)
        m[i] = i/N;
}
